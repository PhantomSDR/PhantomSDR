#include "hip/hip_runtime.h"
#include <cassert>
#include <stdexcept>

#include "fft.h"

cuFFT::cuFFT(size_t size, int nthreads, int downsample_levels, int brightness_offset)
    : FFT(size, nthreads, downsample_levels, brightness_offset), plan{0} {
    int count;
    hipGetDeviceCount(&count);
    if (!count) {

        throw std::runtime_error("No CUDA devices found");
    }

    hipMalloc(&cuda_windowbuf, sizeof(float) * size);
    hipMemcpy(cuda_windowbuf, windowbuf, sizeof(float) * size,
               hipMemcpyHostToDevice);
    operator delete[](windowbuf, std::align_val_t(32));
    windowbuf = NULL;
}

float *cuFFT::malloc(size_t size) {
    float *ptr;
    hipError_t err =
        hipHostAlloc(&ptr, sizeof(float) * size, hipHostMallocMapped);
    return ptr;
}
void cuFFT::free(float *ptr) { hipHostFree(ptr); }

int cuFFT::plan_c2c(direction d, int) {
    assert(!plan);

    hipMalloc(&cuda_inbuf, sizeof(float) * size * 2);
    hipMallocManaged(&outbuf, sizeof(float) * (size + additional_size) * 2);
    cuda_outbuf = outbuf;
    hipMalloc(&cuda_powerbuf, sizeof(float) * size * 2);
    hipMallocManaged(&quantizedbuf, sizeof(int8_t) * size * 2);
    cuda_quantizedbuf = quantizedbuf;
    outbuf_len = size;

    type = HIPFFT_C2C;
    hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);
    cuda_direction = d == FORWARD ? HIPFFT_FORWARD : HIPFFT_BACKWARD;
    return 0;
}
int cuFFT::plan_r2c(int) {
    assert(!plan);

    hipMalloc(&cuda_inbuf, sizeof(float) * size);
    hipMallocManaged(&outbuf, sizeof(float) * (size + 2));
    cuda_outbuf = outbuf;
    hipMalloc(&cuda_powerbuf, sizeof(float) * size * 2);
    hipMallocManaged(&quantizedbuf, sizeof(int8_t) * size * 2);
    cuda_quantizedbuf = quantizedbuf;
    outbuf_len = size / 2;

    type = HIPFFT_R2C;
    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    return 0;
}

__global__ void window_real(float *output, float *input, float *window,
                            size_t len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < len; i += stride)
        output[i] = input[i] * window[i];
}
__global__ void window_complex(float *output, float *input, float *window,
                               size_t len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < len; i += stride) {
        output[i * 2] = input[i * 2] * window[i];
        output[i * 2 + 1] = input[i * 2 + 1] * window[i];
    }
}
int cuFFT::load_real_input(float *a1, float *a2) {
    hipHostGetDevicePointer(&a1, a1, 0);
    hipHostGetDevicePointer(&a2, a2, 0);
    int blockSize = 1024;
    int numBlocks = (size / 2 + blockSize - 1) / blockSize;
    window_real<<<numBlocks, blockSize>>>(cuda_inbuf, a1, cuda_windowbuf,
                                          size / 2);
    window_real<<<numBlocks, blockSize>>>(&cuda_inbuf[size / 2], a2,
                                          &cuda_windowbuf[size / 2], size / 2);
    return 0;
}
int cuFFT::load_complex_input(float *a1, float *a2) {
    hipHostGetDevicePointer(&a1, a1, 0);
    hipHostGetDevicePointer(&a2, a2, 0);
    int blockSize = 1024;
    int numBlocks = (size / 2 + blockSize - 1) / blockSize;
    window_complex<<<numBlocks, blockSize>>>(cuda_inbuf, a1, cuda_windowbuf,
                                             size / 2);
    window_complex<<<numBlocks, blockSize>>>(
        &cuda_inbuf[size], a2, &cuda_windowbuf[size / 2], size / 2);
    return 0;
}

__device__ inline int log_power(float power, int power_offset) {
    return max(-128, __float2int_rz(20 * log10f(power) +
                                    power_offset * 6.020599913279624 + 127));
}
__global__ void power_and_quantize(float *complexbuf, float *powerbuf,
                                   int8_t *quantizedbuf, float normalize,
                                   size_t outbuf_len, int power_offset) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < outbuf_len; i += stride) {
        complexbuf[i * 2] /= normalize;
        complexbuf[i * 2 + 1] /= normalize;
        float re = complexbuf[i * 2];
        float im = complexbuf[i * 2 + 1];
        float power = re * re + im * im;
        powerbuf[i] = power;
        quantizedbuf[i] = log_power(power, power_offset);
    }
}
__global__ void half_and_quantize(float *powerbuf, float *halfbuf,
                                  int8_t *quantizedbuf, size_t outbuf_len,
                                  int power_offset) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < outbuf_len; i += stride) {
        float power = powerbuf[i * 2] + powerbuf[i * 2 + 1];
        halfbuf[i] = power;
        quantizedbuf[i] = log_power(power, power_offset);
    }
}

int cuFFT::execute() {
    if (type == HIPFFT_C2C) {
        hipfftExecC2C(plan, (hipfftComplex *)cuda_inbuf,
                     (hipfftComplex *)cuda_outbuf, HIPFFT_FORWARD);
    } else if (type == HIPFFT_R2C) {
        hipfftExecR2C(plan, (hipfftReal *)cuda_inbuf,
                     (hipfftComplex *)cuda_outbuf);
    } else if (type == HIPFFT_C2R) {
        hipfftExecC2R(plan, (hipfftComplex *)cuda_inbuf,
                     (hipfftReal *)cuda_outbuf);
    }

    int base_idx = 0;
    bool is_complex = outbuf_len == size;
    // For IQ input, the lowest frequency is in the middle
    if (is_complex) {
        base_idx = size / 2 + 1;
    }
    // outbuf is complex so we need to multiply by 2
    int blockSize = 1024;
    int numBlocks = (outbuf_len - base_idx + blockSize - 1) / blockSize;
    power_and_quantize<<<numBlocks, blockSize>>>(
        &cuda_outbuf[base_idx * 2], cuda_powerbuf, cuda_quantizedbuf, size,
        outbuf_len - base_idx, size_log2);

    numBlocks = (base_idx + blockSize - 1) / blockSize;
    power_and_quantize<<<numBlocks, blockSize>>>(
        cuda_outbuf, &cuda_powerbuf[outbuf_len - base_idx],
        &cuda_quantizedbuf[outbuf_len - base_idx], size, base_idx, size_log2);

    int out_len = outbuf_len;
    int8_t *quantized_offset_buf = cuda_quantizedbuf;
    float *power_offset_buf = cuda_powerbuf;
    for (int i = 0; i < downsample_levels - 1; i++) {
        numBlocks = (out_len / 2 + blockSize - 1) / blockSize;
        half_and_quantize<<<numBlocks, blockSize>>>(
            power_offset_buf, power_offset_buf + out_len,
            quantized_offset_buf + out_len, out_len / 2, size_log2 - i - 1);
        power_offset_buf += out_len;
        quantized_offset_buf += out_len;
        out_len /= 2;
    }

    hipDeviceSynchronize();
    return 0;
}
cuFFT::~cuFFT() {
    if (plan) {
        hipfftDestroy(plan);
        hipFree(cuda_inbuf);
        hipFree(cuda_outbuf);
        hipFree(cuda_windowbuf);
        hipFree(cuda_powerbuf);
    }
}